#include "hip/hip_runtime.h"
#include <stdio.h>
#include "timerc.h"

__global__ void cumulative_sum(int *a, int *b) {
	int size_per_block = 2 * blockDim.x;
	int start_address = size_per_block * blockIdx.x;

	for(int s = 1; s <= size_per_block / 2; s *= 2) {
		if(threadIdx.x < blockDim.x / s) {
			a[start_address + 2 * s - 1 + threadIdx.x * s * 2] += a[start_address + s - 1 + threadIdx.x * s * 2];
		}
		__syncthreads();
	}
    
    int mult = 1;
    for(int s = size_per_block / 2; s >= 2; s /= 2) {
        if(threadIdx.x < 2 * mult - 1) {
            a[start_address + s - 1 + (s / 2) + threadIdx.x * s] += a[start_address + s - 1 + threadIdx.x * s];
        }
        __syncthreads();
        mult *= 2;
    }
    
    if(threadIdx.x == 0) {
        b[blockIdx.x] = a[start_address + blockDim.x];
    }
}

__global__ void fix_sum(int *a, int *b, int size_per_block) {
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    if(id > size_per_block) {
        a[id] = a[id] + b[blockIdx.x - 1];
    }
    
}

int main() {
    
    int SIZE = 32;
	int *device_input;
	int *host_input = (int *) malloc(SIZE * sizeof(int));
    int *host_output = (int *) malloc(SIZE * sizeof(int));
    int threads_per_block = 16;
    int size_per_block = 2 * threads_per_block;
    int num_blocks = (SIZE + size_per_block - 1) / size_per_block;
    
	float time;


	hipMalloc(&device_input, SIZE * sizeof(int));

	for(int i = 0; i < SIZE; i++) {
		host_input[i] = 1;
	}

    hipMemcpy(device_input, host_input, SIZE * sizeof(int), hipMemcpyHostToDevice);

	gstart();
	cumulative_sum<<<num_blocks, threads_per_block>>>(device_input);

	hipMemcpy(host_output, device_input, SIZE * sizeof(int), hipMemcpyDeviceToHost);
	gend(&time);

	printf("it took %f seconds\n", time);


	
	for(int i = 0; i < SIZE; i++) {
        printf("%d ", host_output[i]);
	/*	int correct = (i * (i + 1)) / 2;
		if(a_host[i] != correct) 
			printf("Error at pos: %d expected: %d actual: %d\n", i, correct, a_host[i]); */
	}
    printf("\n");
    
    free(host_output);
    free(host_input);
    hipFree(device_input);
    
    return 0;
}
